
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000
#define LD 16
const int threadsPerBlock = LD * LD;

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void kernel(int* a, int* b, int* c_partial){
    __shared__ float cache[threadsPerBlock];
    int flatBlockIdx = blockIdx.x + blockIdx.y * gridDim.x;
    int flatThreadIdx = threadIdx.x + threadIdx.y * blockDim.x;
    int idx = flatThreadIdx + flatBlockIdx * blockDim.x * blockDim.y;
    
    float temp = 0;
    if(idx < N){
        temp += a[idx] * b[idx];
        idx += blockDim.x * blockDim.y * gridDim.x * gridDim.y;
    }

    // writing to cache can be done independently bc each thread has its own space to write
    cache[flatThreadIdx] = temp;

    // yg perlu ditunggu tuh kalo udah beres write to cache semua
    __syncthreads();

    // now, reduce operation.
    // array cache tadi dilipet2 ampe jadi satu (definisi lipet: diadd biasa aja)    
    int middle = threadsPerBlock/2;
    while (middle != 0) {
        if(flatThreadIdx < middle){
            cache[flatThreadIdx] += cache[flatThreadIdx+middle];
        }
        __syncthreads(); // make sure every thread has ngelipet their portion already
        
        // uncomment if u wanna see the addition reduce process wkwk
        // if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
        //     for(int i = 0; i < middle ; i++){
        //         printf("%.0f ", cache[i]);
        //     }
        //     printf("\n");
        // }

        middle /= 2;
    }

    // now put the cache[0] (result of the reduce, local to the block)
    // in the c_partial that that's the size of the number of blocks that we have
    // satu thread aja yang ngelakuin wkoakwawok
    if (flatThreadIdx == 0) {
        c_partial[flatBlockIdx] = cache[0];
    }
}

int main(void){
    // for dot product, c here is a intermediate value for each block
    // we're utilizing device shared memory that's local to every block
    // therefore every thread within a block yagesya
    int *a, *b, *c_partial;
    int *dev_a, *dev_b, *dev_c_partial;

    // let's use the same 2D block and 2D threads format
    // bc hard times create strong men
    dim3 numBlocks(LD, LD, 1);
    dim3 threadsPerBlock(LD, LD, 1);

    // malloc biasa ygy
    a = (int*) malloc(sizeof(int) * N);
    b = (int*) malloc(sizeof(int) * N);
    // cache perlu dialloc ga kalo gitu?
    // cache (and dev_c_partial) will be an array yg masih perlu disum (hasil dari tiap block)
    // but it's gonna be small enough that it's reasonable to compute sequentially
    // but how long is it? it's supposed to be as long as how many blocks we have
    int totalBlocks = numBlocks.x * numBlocks.y * numBlocks.z;
    c_partial = (int*) malloc(sizeof(int) * totalBlocks); 

    // filling the array
    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i * 2;
    }

    // cudamalloc
    HANDLE_ERROR(hipMalloc((void**) &dev_a, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void**) &dev_b, sizeof(int) * N));
    HANDLE_ERROR(hipMalloc((void**) &dev_c_partial, sizeof(int) * totalBlocks));

    // copy input to device
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

    kernel<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_c_partial);

    // copy output back to host
    HANDLE_ERROR(hipMemcpy(c_partial, dev_c_partial, sizeof(int) * totalBlocks, hipMemcpyDeviceToHost));

    float c = 0;
    for(int i = 0; i < totalBlocks; i++){
        c += c_partial[i];
    }

    printf("dot product : %.2f\n", c);

    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c_partial));

    free(a);
    free(b);
    free(c_partial);

    return 0;
}
